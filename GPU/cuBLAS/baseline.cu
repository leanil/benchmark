#include "util.h"
#include "hipblas.h"
#include "hip/hip_runtime_api.h"
#include <iostream>

using namespace std;

void baseline_sum(hipblasHandle_t handle) {
    auto A = init<i_1d>();
    scl_t a;
    checkCudaErrors(hipblasSasum(handle, i_1d, A, 1, &a));
    cout << a << endl;
}

void baseline_inc(hipblasHandle_t handle) {
    auto A = init<i_1d>();
    auto c1 = init<i_1d>(CONST, 1);
    scl_t alpha = 1;
    checkCudaErrors(hipblasSaxpy(handle, i_1d, &alpha, c1, 1, A, 1));
    cout << *A.tmp_data << ' ';
    checkCudaErrors(hipblasGetVector(i_1d, sizeof(scl_t), A.gpu_data, 1, A.tmp_data, 1));
    cout << *A.tmp_data << endl;
}

void baseline_prod(hipblasHandle_t handle) {
    auto A = init<i_1d>();
    scl_t c = 3.141592;
    checkCudaErrors(hipblasSscal(handle, i_1d, &c, A, 1));
    checkCudaErrors(hipblasGetVector(i_1d, sizeof(scl_t), A.gpu_data, 1, A.tmp_data, 1));
    cout << *A.tmp_data << endl;
}
